#include "hip/hip_runtime.h"
/*using namespace std;*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include "kokrusher/cuboard.h"
extern "C" {
#include "board.h"
}
#include "kokrusher/cudamst.h"

#define MAX_PLAYS 400
#define N_PLAYOUTS 100

#define CUDA_CALL(x) __checkerr((x), __FILE__, __LINE__) 

#define cudaAllocDevArray(dp_,size_) do { \
    void *lptr = NULL; \
    CUDA_CALL(hipMalloc(&lptr, size_)); \
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dp_), &lptr, sizeof(void *))); \
    } while(0)

//Global board state array variables.
//Yes this code is horrible. I'm sorry.
__device__ hiprandState randStates[M*N];
__device__ int g_flen[M*N];
__device__ stone (*g_b)[M*N];
__device__ coord_t (*g_f)[M*N];
__device__ coord_t (*g_p)[M*N];
__device__ group_t (*g_g)[M*N];
__device__ int (*g_libs)[M*N];
__device__ coord_t (*g_gi)[GROUP_KEEP_LIBS][M*N];
__device__ int g_caps[S_MAX][M*N];
__device__ char (*g_ncol)[S_MAX][M*N];

// this method roughly corresponds to the Simulate method from the pseudocode
__global__ void run_sims(enum stone color, float *votes, int moves, int passes, int size, void *data){
    /*cuboard_init(size);*/
    hiprandState myState = randStates[bid];
    float win = 0;
    int first_move = bid % my_flen;
    int i;
    //where uct happens: SimTree method from pseudocode
    for (i=0; i<N_PLAYOUTS; i++) {
        cuboard_copy(data, size);
        cuboard_play(color, first_move, size);
        win += cuda_play_random_game(color, myState, moves, passes, size);
    }
    atomicAdd(&votes[first_move], win/N_PLAYOUTS);
    //record move: Backup method from pseudocode
}

// this method corresponds to SimDefault in the pseudocode
__device__ float 
cuda_play_random_game(enum stone starting_color, hiprandState rState, int moves, int passes, int size)
{
	int gamelen = MAX_PLAYS - moves;
	enum stone color = starting_color;
	while (gamelen-- && passes < 2) {
		color = custone_other(color);
        coord_t coord;
		cuboard_play_random(color, &coord, rState, size);
		if (IS_PASS(coord)) {
			passes++;
		} else {
			passes = 0;
		}
	}
	float score = cuboard_fast_score(size);
	return (starting_color == S_WHITE ^ score < 0? 1 : -1);
}

// this going to correspond to the UctSearch method in the pseudocode from the literature
coord_t *cuda_genmove(struct board *b, struct time_info *ti, enum stone color){
    float *votes = NULL, *hVotes=NULL;
    size_t vote_size = b->flen * sizeof(float);
	int passes = IS_PASS(b->last_move.coord) && b->moves > 0;
    void *data = NULL, *hData = NULL;
    int data_size = copy_essential_board_data(b, &hData);
    printf("malloced data %d bytes\n", data_size);
    assert(hData != NULL);
    
    //allocate vote array
    hVotes = (float *) malloc(vote_size);
    CUDA_CALL(hipMalloc(&votes, vote_size));
    CUDA_CALL(hipMemset(votes, 0, vote_size));

    //allocate and copy board data
    CUDA_CALL(hipMalloc(&data, data_size));
    CUDA_CALL(hipMemcpy(data, hData, data_size, hipMemcpyHostToDevice));
    free(hData);

    run_sims<<<M,N>>>(color, votes, b->moves, passes, b->size, data);
    CUDA_CALL(hipPeekAtLastError());

    CUDA_CALL(hipMemcpy(hVotes, votes, vote_size, hipMemcpyDeviceToHost));
    CUDA_CALL(hipFree(votes));
    CUDA_CALL(hipFree(data));

    coord_t *my_move = (coord_t *) malloc(sizeof(coord_t));
    *my_move=-1;
    int i;
    for (i=0; i<b->flen; i++) {
        printf("vote  %d=%d c=%d\n", i, hVotes[i], b->f[i]);
        if (hVotes[i] > *my_move) {
            *my_move = b->f[i];
        }
    }
    free(hVotes);
    return my_move;
}

void init_kokrusher(struct board *b){
    size_t size = b->size * b->size;

    cudaAllocDevArray(g_b, sizeof(enum stone) * M*N * size);
    cudaAllocDevArray(g_p, sizeof(coord_t) * M*N * size);
    cudaAllocDevArray(g_g, sizeof(group_t) * M*N * size);
    cudaAllocDevArray(g_libs, sizeof(int) * M*N * size);
    cudaAllocDevArray(g_gi, sizeof(coord_t) * GROUP_KEEP_LIBS * M*N * size);
    cudaAllocDevArray(g_ncol, sizeof(char) * S_MAX * M*N * size);
    cudaAllocDevArray(g_f, sizeof(coord_t) * M*N * size);

    //initialize random states
    cuda_rand_init<<<M,N>>>(time(NULL));    
    CUDA_CALL(hipPeekAtLastError());

    printf("engine initialized\n");
    return;
}

size_t copy_essential_board_data(struct board * b, void **d){
    int size2 = board_size2(b);
    int bsize = size2 * sizeof(*b->b);
	int fsize = size2 * sizeof(*b->f);
	int psize = size2 * sizeof(*b->p);
	int gsize = size2 * sizeof(*b->g);
    int glibsize = sizeof(int) * size2;
    int gisize = sizeof(int) * size2 * GROUP_KEEP_LIBS;
    int ncolsize = sizeof(int) * size2 * S_MAX;
    int capsize = sizeof(int) * S_MAX;
    int flensize = sizeof(int);
    int total = bsize + fsize + psize + gsize + glibsize + gisize + ncolsize + capsize + flensize;
    int i,j;
    char *data = (char *) malloc(total);
    *d = (void *) data;
    memcpy(data, b->b, bsize);
    data += bsize;
    memcpy(data, b->f, fsize);
    data += fsize;
    memcpy(data, b->p, psize);
    data += psize;
    memcpy(data, b->g, gsize);
    data += gsize;
    for (i=0; i<size2; i++)
        ((int *) data)[i] = b->gi[i].libs;
    data += glibsize;
    for (i=0; i<size2; i++)
        for (j=0; j<GROUP_KEEP_LIBS; j++)
            ((int *) data)[(i*GROUP_KEEP_LIBS) + j] = b->gi[i].lib[j];
    data += gisize;
    for (i=0; i<size2; i++)
        for (j=0; j<S_MAX; j++)
            ((int *) data)[(i*S_MAX) + j] = b->n[i].colors[j];
    data += ncolsize;
    for (i=0; i<S_MAX; i++)
        ((int *) data)[i] = b->captures[i];
    data += capsize;
    *((int *) data) = b->flen;
    return total;
}

void __checkerr(hipError_t e, char * file, int line){
    if (e != hipSuccess){
      printf("Error %s at %s:%d\n",hipGetErrorString(e),file,line); 
      exit(1);
    }
}

__global__ void cuda_rand_init(unsigned long seed){
    int id = threadIdx.x;
    hiprand_init(seed, id, 0, randStates + id);
}
