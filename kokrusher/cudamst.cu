#include "hip/hip_runtime.h"
/*using namespace std;*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include "kokrusher/cuboard.h"
#include "kokrusher/uct.h"
extern "C" {
#include "board.h"
}
#include "kokrusher/cudamst.h"

#define MAX_PLAYS 400
#define N_PLAYOUTS 30

#define CUDA_CALL(x) __checkerr((x), __FILE__, __LINE__) 

#define cudaAllocDevArray(dp_,size_) do { \
    void *lptr = NULL; \
    CUDA_CALL(hipMalloc(&lptr, size_)); \
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dp_), &lptr, sizeof(void *))); \
    } while(0)

//Global board state array variables.
//Yes this code is horrible. I'm sorry.
__device__ hiprandState randStates[M*N];
__device__ int g_flen[M*N];
__device__ int next_alloc;
__device__ stone (*g_b)[M*N];
__device__ int (*g_tree)[4];
__device__ coord_t (*g_f)[M*N];
__device__ coord_t (*g_p)[M*N];
__device__ group_t (*g_g)[M*N];
__device__ int (*g_libs)[M*N];
__device__ unsigned char (*g_watermark)[M*N];
__device__ coord_t (*g_gi)[GROUP_KEEP_LIBS][M*N];
__device__ int g_caps[S_MAX][M*N];
__device__ char (*g_ncol)[S_MAX][M*N];

__global__ void reset_tree(enum stone color, int size, void *data){
    //call with one block of n threads where n is the number of theoretically board positions
    //ei 81 for a 9X9 board
    if (threadIdx.x == 0){
        init_tree(0, -1);
        tree_child(0) = 1;
        tree_visits(0) = 1;
        next_alloc = 1 + child_size;
    }
    cuboard_copy(data, size);
    int i = threadIdx.x + 1;
    init_tree(i, 0);
    if (cuboard_is_valid_play(color, index_2_coord(threadIdx.x, size), size)) {
        tree_visits(i) = 0;
    }
}

__global__ void harvest_data(int* data, int size) {
    *data = best_move(size);
}

// this method roughly corresponds to the Simulate method from the pseudocode
__global__ void run_sims(enum stone color, int moves, int passes,float offset, int size, void *data){
    /*cuboard_init(size);*/
    hiprandState myState = randStates[bid];
    enum stone cur_col = color;
    int win;
    int i;
    //where uct happens: SimTree method from pseudocode
    for (i=0; i<N_PLAYOUTS; i++) {
        cuboard_copy(data, size);
        /*int node = walk_down(&cur_col, size);*/
        float score  = cuda_play_random_game(color, myState, moves, passes, size) + offset;
	    win = ((color == S_WHITE) ^ (score < 0));
        /*backup(node, win);*/
    }
    //record move: Backup method from pseudocode
}

// this method corresponds to SimDefault in the pseudocode
__device__ int 
cuda_play_random_game(enum stone starting_color, hiprandState rState, int moves, int passes, int size)
{
	int gamelen = MAX_PLAYS - moves;
	enum stone color = starting_color;
	while (gamelen-- && passes < 2) {
        coord_t coord;
		cuboard_play_random(color, &coord, rState, size);
		if (IS_PASS(coord)) {
			passes++;
		} else {
			passes = 0;
		}
		color = custone_other(color);
	}
	return cuboard_fast_score(size);
}

// this going to correspond to the UctSearch method in the pseudocode from the literature
coord_t *cuda_genmove(struct board *b, struct time_info *ti, enum stone color){
	int passes = IS_PASS(b->last_move.coord) && b->moves > 0;
    void *data = NULL, *hData = NULL;
    int data_size = copy_essential_board_data(b, &hData);
    float offset = b->komi + b->handicap;
    assert(hData != NULL);

    //allocate and copy board data
    CUDA_CALL(hipMalloc(&data, data_size));
    CUDA_CALL(hipMemcpy(data, hData, data_size, hipMemcpyHostToDevice));
    free(hData);

    reset_tree<<<1,(b->size-2)*(b->size-2)>>>(color,b->size, data);
    CUDA_CALL(hipDeviceSynchronize());
    printf("Reset Tree\n");

    run_sims<<<M,N>>>(color, b->moves, passes, offset, b->size, data);
    CUDA_CALL(hipPeekAtLastError());

    coord_t * best_move;
    coord_t *my_move = (coord_t *) malloc(sizeof(coord_t));

    CUDA_CALL(hipMalloc(&best_move, sizeof(coord_t)));
    harvest_data<<<1,1>>>(best_move, b->size);

    CUDA_CALL(hipDeviceSynchronize());
    printf("Collected Results\n");

    CUDA_CALL(hipMemcpy(my_move, best_move, sizeof(coord_t), hipMemcpyDeviceToHost));
    printf("Playing %d\n", *my_move);
    return my_move;
}

void init_kokrusher(struct board *b){
    size_t size = b->size * b->size;
    /*CUDA_CALL(cudaThreadSetLimit(hipLimitMallocHeapSize, heap_size));*/
    cudaAllocDevArray(g_watermark, (M*N*(size/8) * sizeof(unsigned char)));
    cudaAllocDevArray(g_b, sizeof(enum stone) * M*N * size);
    cudaAllocDevArray(g_p, sizeof(coord_t) * M*N * size);
    cudaAllocDevArray(g_g, sizeof(group_t) * M*N * size);
    cudaAllocDevArray(g_libs, sizeof(int) * M*N * size);
    cudaAllocDevArray(g_gi, sizeof(coord_t) * GROUP_KEEP_LIBS * M*N * size);
    cudaAllocDevArray(g_ncol, sizeof(char) * S_MAX * M*N * size);
    cudaAllocDevArray(g_f, sizeof(coord_t) * M*N * size);
    cudaAllocDevArray(g_tree, sizeof(int) * 4 * MAX_ALLOC);

    //initialize random states
    cuda_rand_init<<<M,N>>>(time(NULL));    
    CUDA_CALL(hipPeekAtLastError());

    printf("engine initialized\n");
    return;
}

size_t copy_essential_board_data(struct board * b, void **d){
    int size2 = board_size2(b);
    int bsize = size2 * sizeof(*b->b);
	int fsize = size2 * sizeof(*b->f);
	int psize = size2 * sizeof(*b->p);
	int gsize = size2 * sizeof(*b->g);
    int glibsize = sizeof(int) * size2;
    int gisize = sizeof(int) * size2 * GROUP_KEEP_LIBS;
    int ncolsize = sizeof(int) * size2 * S_MAX;
    int capsize = sizeof(int) * S_MAX;
    int flensize = sizeof(int);
    int total = bsize + fsize + psize + gsize + glibsize + gisize + ncolsize + capsize + flensize;
    int i,j;
    char *data = (char *) malloc(total);
    *d = (void *) data;
    memcpy(data, b->b, bsize);
    data += bsize;
    memcpy(data, b->f, fsize);
    data += fsize;
    memcpy(data, b->p, psize);
    data += psize;
    memcpy(data, b->g, gsize);
    data += gsize;
    for (i=0; i<size2; i++)
        ((int *) data)[i] = b->gi[i].libs;
    data += glibsize;
    for (i=0; i<size2; i++)
        for (j=0; j<GROUP_KEEP_LIBS; j++)
            ((int *) data)[(i*GROUP_KEEP_LIBS) + j] = b->gi[i].lib[j];
    data += gisize;
    for (i=0; i<size2; i++)
        for (j=0; j<S_MAX; j++)
            ((int *) data)[(i*S_MAX) + j] = b->n[i].colors[j];
    data += ncolsize;
    for (i=0; i<S_MAX; i++)
        ((int *) data)[i] = b->captures[i];
    data += capsize;
    *((int *) data) = b->flen;
    return total;
}

void __checkerr(hipError_t e, char * file, int line){
    if (e != hipSuccess){
      printf("Error %s at %s:%d\n",hipGetErrorString(e),file,line); 
      exit(1);
    }
}

__global__ void cuda_rand_init(unsigned long seed){
    hiprand_init(seed, bid, 0, &randStates[bid]);
}
